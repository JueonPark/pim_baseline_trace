
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

/**
 * random number generation for matrix-vector
 **/
__global__ void generate_matrix_vector(int row, int col) {

}

/**
 * Simple matrix-vector-multiplication
 * C: result
 * A: 
 **/
__global__ void mv_multiply( float* C, float* A, float* B, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0.0f;
  for (int k = 0; k < n; k++) {
    sum += A[row*n+k] * B[k * n + col];
  }
  C[row*n+col] = sum;
}

/**
 * arguments: row, column
 * matrix-vector multiplication of (r*c) and c
 **/
int main(int argc, char** argv) {
  std::cout << argc << "\n";
  if (argc != 3) {
    assert(0 && "matrix-vector multiplication requires row and column argument");
  }
  int row = std::stoi(argv[1]);
  int col = std::stoi(argv[2]);

  float* a = (float*)malloc(row*col*sizeof(float));
  float* b = (float*)malloc(col*sizeof(float));
  float* c = (float*)malloc(row*sizeof(float));
  
  float *A, *B, *C;
  cublasInit();
  cublasAlloc(row * col, sizeof(float), (void**) &A);
  cublasAlloc(col * 1, sizeof(float), (void**) &B);
  cublasAlloc(row * 1, sizeof(float), (void**) &C);

  hipblasSetMatrix(row, col, sizeof(float*), a, col, A, col);
  hipblasSetVector(col, sizeof(float), b, 1, B, 1);

  float alpha = 1.0;
  float beta = 0.0;

  cublasSgemv('N', row, col, alpha, A, row, B, 1, beta, C, 1);

  return 0;
}
